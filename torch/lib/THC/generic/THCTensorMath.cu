
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorMath.cu"
#else

THC_API void
THCTensor_(fill)(THCState* state, THCTensor *self_, real value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));

  if (!THC_pointwiseApply1(
        state, self_, TensorFillOp<real>(value))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zero)(THCState *state, THCTensor *self_)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  if (THCTensor_(isContiguous)(state, self_)) {
    THCudaCheck(cudaMemsetAsync(THCTensor_(data)(state, self_),
                                0,
                                sizeof(real) * THCTensor_(nElement)(state, self_),
                                THCState_getCurrentStream(state)));
  } else {
    if (!THC_pointwiseApply1(
          state, self_,
          TensorFillOp<real>(ScalarConvert<int, real>::to(0)))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

THC_API void
THCTensor_(zeros)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(zero)(state, r_);
}

THC_API void
THCTensor_(ones)(THCState *state, THCTensor *r_, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(fill)(state, r_, ScalarConvert<int, real>::to(1));
}

THC_API void
THCTensor_(reshape)(THCState *state, THCTensor *r_, THCTensor *t, THLongStorage *size)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, r_, t));
  THCTensor_(resize)(state, r_, size, NULL);
  THCTensor_(copy)(state, r_, t);
}

ptrdiff_t
THCTensor_(numel)(THCState *state, THCTensor *t)
{
  return THCTensor_(nElement)(state, t);
}

void THCTensor_(cat)(THCState *state, THCTensor *result,
		     THCTensor *ta, THCTensor *tb, int dimension)
{
  THCTensor* inputs[2];
  inputs[0] = ta;
  inputs[1] = tb;
  THCTensor_(catArray)(state, result, inputs, 2, dimension);
}

void THCTensor_(catArray)(THCState *state, THCTensor *result,
			  THCTensor **inputs, int numInputs, int dimension)
{
  THLongStorage *size;
  int i, j, cohortMax;
  int64_t offset;
  bool hasEmptyInput = false;

  // Even in the case where dimension is negative (i.e. when we want
  // to cat along the last dimension), this logic still works, as the
  // loop below will overwrite the value
  int maxDim = dimension + 1;

  // cat_dimension is the actual dimension we cat along
  int cat_dimension = dimension;

  for (i = 0; i < numInputs; i++)
  {
    int inputDim = THCTensor_(nDimension)(state, inputs[i]);
    hasEmptyInput |= !inputDim;
    maxDim = THMax(maxDim, inputDim);
  }

  // In the event that the user specified -1 as the concat dimension, then
  // we want to pick the maxDim  as dimension to cat along (and thus maxDim - 1 as the
  // value due to 0-based indexing). If the maxDim is // 0 (i.e. we are catting all
  // empty tensors), then we set cat_dimension to be 0
  if (dimension + TH_INDEX_BASE == -1) {
    cat_dimension = maxDim ? (maxDim - 1) : 0;
  }

  THArgCheck(numInputs > 0, 3, "invalid number of inputs %d", numInputs);
  THArgCheck(cat_dimension >= 0, 4, "invalid dimension %d", dimension + TH_INDEX_BASE);

  size = THLongStorage_newWithSize(maxDim);
  for(i = 0; i < maxDim; i++)
  {
    // dimSize is either the size of the dim if it exists, either 1 if #dim > 0, otherwise 0
    int64_t dimSize = i < THCTensor_(nDimension)(state, inputs[0])
                       ? THCTensor_(size)(state, inputs[0], i)
                       : THMin(THCTensor_(nDimension)(state, inputs[0]), 1);
    if (i == cat_dimension)
    {
      for (j = 1; j < numInputs; j++)
      {
        // accumulate the size over the dimension we want to cat on.
        // Empty tensors are allowed
        dimSize += i < THCTensor_(nDimension)(state, inputs[j])
                       ? THCTensor_(size)(state, inputs[j], i)
                       : THMin(THCTensor_(nDimension)(state, inputs[j]), 1);
      }
    }
    else
    {
      for (j = 1; j < numInputs; j++)
      {
        int64_t sz = i < THCTensor_(nDimension)(state, inputs[j])
                      ? THCTensor_(size)(state, inputs[j], i)
                      : THMin(THCTensor_(nDimension)(state, inputs[j]), 1);

        // If it's a dimension we're not catting on
        // Then fail if sizes are different AND > 0
        if (dimSize != sz && dimSize && sz) {
          THLongStorage_free(size);
          THError("inconsistent tensor sizes");
        }
        else if(!dimSize)
        {
          dimSize = sz;
        }
      }
    }
    size->data[i] = dimSize;
  }

  THCTensor_(resize)(state, result, size, NULL);
  THLongStorage_free(size);

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. No empty inputs
  // 3. The result tensor is 32-bit indexable
  // 4. The number of dimensions is <= 4
  // 5. All input tensors are contiguous (output tensor may be non-contig)
  // 6. All input tensors can use 32-bit indexing
  // 7. All input tensors are on the same device

  if (numInputs > 1 &&
      !hasEmptyInput &&
      THCTensor_(nDimension)(state, result) <= CAT_ARRAY_MAX_INPUT_DIMS &&
      TensorUtils<THCTensor>::canUse32BitIndexMath(state, result) &&
      TensorUtils<THCTensor>::allContiguous(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::all32BitIndexable(state, inputs, numInputs) &&
      TensorUtils<THCTensor>::allSameDevice(state, inputs, numInputs)) {

    // First, let's set up our kernel parameters. We start with a raw pointer to the storage
    // for the output Tensor.
    real *data = THCTensor_(data)(state, result);

    // Kernel Parameter
    CatArrInputTensor<real, unsigned int> stackInputs[CAT_ARRAY_BATCH_SIZE];
    CatArrInputTensor<real, unsigned int> *d_inputs;

    // Attempt to re-use stream's scratch space for the input metadata
    bool usedScratch = false;
    size_t tensorMetadataSize = sizeof(CatArrInputTensor<real, unsigned int>) * CAT_ARRAY_BATCH_SIZE;
    if (THCState_getCurrentDeviceScratchSpaceSize(state) > tensorMetadataSize) {
      void* space = THCState_getCurrentDeviceScratchSpace(state);
      if (space) {
        d_inputs = (CatArrInputTensor<real, unsigned int> *) space;
        usedScratch = true;
      }
    }
    if (!usedScratch) {
      // Fallback to allocating GPU memory
      THCudaCheck(THCudaMalloc(state, (void**) &d_inputs, tensorMetadataSize));
    }

    OutputTensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> param;

    // Next, let's initialize the size, stride arrays for the output Tensor.
    for (i = 0; i < maxDim; ++i) {
      param.outputSize[i] = THCTensor_(size)(state, result, i);
      param.outputStride[i] = THCTensor_(stride)(state, result, i);
    }

    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
  CatArrayBatchedCopy<real, unsigned int, DIMS><<<applyGrid, applyBlock>>>(data, d_inputs, param, cat_dimension, param.outputStride[cat_dimension]);

    // Now we loop
    offset = 0;
    for (i = 0; i < numInputs; i += CAT_ARRAY_BATCH_SIZE) {
      cohortMax = 0;
      for (j = 0; j < CAT_ARRAY_BATCH_SIZE && (i+j) < numInputs; ++j) {
        int64_t dimSize = cat_dimension < THCTensor_(nDimension)(state, inputs[i+j])
          ? THCTensor_(size)(state, inputs[i+j], cat_dimension)
          : 1;

        stackInputs[j].input = THCTensor_(data)(state, inputs[i+j]);
        stackInputs[j].offset = offset;
        stackInputs[j].dimSize = dimSize;
        stackInputs[j].nElements = THCTensor_(nElement)(state, inputs[i+j]);
        cohortMax = cohortMax > stackInputs[j].nElements ? cohortMax : stackInputs[j].nElements;

        // update offset
        offset += dimSize;
      }
      THCudaCheck(cudaMemcpy(d_inputs, stackInputs, j * sizeof(CatArrInputTensor<real, unsigned int>), cudaMemcpyHostToDevice));

      // Next, let's consider how we set our kernel launch parameters.
      // We borrow from THCApply, which the kernel's internal indexing
      // is based on.
      dim3 applyBlock = getApplyBlock();

      // We also re-use the applyGrid - but note that we use the maximum number of
      // elements for a given tensor in this grouping to determine the count
      dim3 applyGrid;
      getApplyGrid(state, cohortMax, applyGrid);

      // Next, we set our grid's y component to be the number of tensors in
      // the batch. This will allow the kernel to determine which input
      // tensor it is responsible for copying
      applyGrid.y = j;

      switch (maxDim) {
        case 1:
          HANDLE_CASE(1);
          break;
        case 2:
          HANDLE_CASE(2);
          break;
        case 3:
          HANDLE_CASE(3);
          break;
        case 4:
          HANDLE_CASE(4);
          break;
      }
      THCudaCheck(cudaGetLastError());
    }
    if (!usedScratch) {
      THCudaCheck(THCudaFree(state, (void *)d_inputs));
    }
#undef HANDLE_CASE
  } else {
    offset = 0;
    for (j = 0; j < numInputs; j++)
    {
      // No reason to copy when input is empty
      if (!THCTensor_(nDimension)(state, inputs[j])) continue;

      int64_t dimSize = cat_dimension < THCTensor_(nDimension)(state, inputs[j])
               ? THCTensor_(size)(state, inputs[j], cat_dimension)
               : 1;

      THCTensor *nt = THCTensor_(newWithTensor)(state, result);
      THCTensor_(narrow)(state, nt, NULL, cat_dimension, offset, dimSize);
      THCTensor_(copy)(state, nt, inputs[j]);
      THCTensor_(free)(state, nt);
      offset += dimSize;
    }
  }
}

void THCTensor_(nonzero)(THCState* state, THCudaLongTensor *tensor,
                          THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self  ));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, tensor));


  using namespace thrust::placeholders;
  THCThrustAllocator thrustAlloc(state);
  self = THCTensor_(newContiguous)(state, self);
  thrust::device_ptr<real> self_data(THCTensor_(data)(state, self));

  int num_dim = THCTensor_(nDimension)(state, self);
  int64_t N = THCTensor_(nElement)(state, self);

  THCudaLongTensor_resize2d(state, tensor, N, num_dim);
  tensor = THCudaLongTensor_newContiguous(state, tensor);
  thrust::device_ptr<int64_t> tensor_data(THCudaLongTensor_data(state, tensor));

  thrust::counting_iterator<int64_t> idxfirst(0);
  thrust::counting_iterator<int64_t> idxlast = idxfirst + N;

  typedef thrust::device_ptr<int64_t> Iter;
  strided_range<Iter> strided_tensor(tensor_data,
                                     tensor_data+N*num_dim, num_dim);

#if CUDA_VERSION >= 7000
  cudaStream_t stream = THCState_getCurrentStream(state);
#endif

  strided_range<Iter>::iterator dend = thrust::copy_if(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(stream),
#endif
    idxfirst,
    idxlast,
    self_data,
    strided_tensor.begin(),
    NonZeroOp<real>()
  );

  int64_t num_nonzeros = thrust::distance(strided_tensor.begin(), dend);

  int64_t div = 1;
  for (int dim = num_dim-1; dim >= 0; dim--) {
    strided_range<Iter> stride_dim(tensor_data+dim,
                                   tensor_data+N*num_dim, num_dim);
    thrust::transform(
#if CUDA_VERSION >= 7000
      thrust::cuda::par(thrustAlloc).on(stream),
#endif
      strided_tensor.begin(),
      strided_tensor.end(),
      stride_dim.begin(),
      idx_functor(div, self->size[dim])
    );
    div *= self->size[dim];
  }

  THCudaLongTensor_resize2d(state, tensor, num_nonzeros, num_dim);

  THCTensor_(free)(state, self);
  THCudaLongTensor_free(state, tensor);

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(diag)(THCState *state, THCTensor *self_, THCTensor *src_, int64_t k){
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  int nDimension = THCTensor_(nDimension)(state, src_);
  THArgCheck((nDimension == 2) || (nDimension == 1), 1, "expected a matrix or a vector");
  if (nDimension == 2) {
    int64_t stride0 = THCTensor_(stride)(state, src_, 0);
    int64_t stride1 = THCTensor_(stride)(state, src_, 1);
    int64_t size0 = THCTensor_(size)(state, src_, 0);
    int64_t size1 = THCTensor_(size)(state, src_, 1);
    int64_t size = (k > 0) ? min((int64_t)size0, (int64_t)size1 - k) : min((int64_t)size0 + k, (int64_t)size1);
    THCTensor_(resize1d)(state, self_, size);
    int64_t strideSelf = THCTensor_(stride)(state, self_, 0);
    const dim3 threads(min((int64_t)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (int64_t)size));
    dim3 grid(min((int64_t)1024, (int64_t)THCCeilDiv(size, (int64_t)threads.x)));
    int64_t start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyFromDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, size, stride0 + stride1, strideSelf);
  } else {
    ptrdiff_t totalElements = THCTensor_(nElement)(state, src_);
    ptrdiff_t size = (k > 0) ? totalElements + k : totalElements - k;
    int64_t strideSrc = THCTensor_(stride)(state, src_, 0);
    THCTensor_(resize2d)(state, self_, size, size);
    THCTensor_(zero)(state, self_);
    int64_t stride0 = THCTensor_(stride)(state, self_, 0);
    int64_t stride1 = THCTensor_(stride)(state, self_, 1);
    const dim3 threads(min((int64_t)THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock, (int64_t)size));
    dim3 grid(min((int64_t)1024, (int64_t)THCCeilDiv(size, (ptrdiff_t)threads.x)));
    ptrdiff_t start = (k >= 0 ? k * stride1 : -k * stride0);
    THCTensor_copyToDiagonal<real><<<grid, threads, 0, THCState_getCurrentStream(state)>>>
    (THCTensor_(data)(state, self_), THCTensor_(data)(state, src_), start, totalElements, stride0 + stride1, strideSrc);
  }
  THCudaCheck(cudaGetLastError());
}

accreal THCTensor_(trace)(THCState *state, THCTensor *src_) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, src_));
  THArgCheck((src_->nDimension == 2), 1, "expected a matrix");
  THCTensor *diag = THCTensor_(new)(state);
  THCTensor_(diag)(state, diag, src_, 0);
  accreal trace = THCTensor_(sumall)(state, diag);
  THCTensor_(free)(state, diag);
  return trace;
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(linspace)(THCState *state, THCTensor *r_, real a, real b, int64_t n) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(n > 1 || (n == 1 && (a == b)), 3, "invalid number of points");
  if (THCTensor_(nElement)(state, r_) != n) THCTensor_(resize1d)(state, r_, n);
  if (n == 1) THCTensor_(fill)(state, r_, a);
  else {
    THCTensor *r = THCTensor_(isContiguous)(state, r_)
                   ? r_ // if r_ is contiguous we can direct work on it
                   : THCTensor_(newContiguous)(state, r_);
    real step = THCNumerics<real>::div(THCNumerics<real>::sub(b, a),
                                       ScalarConvert<int64_t,real>::to(n - 1));
    LinspaceOp<real> linspace_method(a, step);
    thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
    thrust::tabulate(data_, data_ + n, linspace_method);
    if (!THCTensor_(isContiguous)(state, r_)) { // We need to move data back to r_
      THCTensor_(freeCopyTo)(state, r, r_);
    }
  }
  THCudaCheck(cudaGetLastError());
}

void THCTensor_(logspace)(THCState *state, THCTensor *r_, real a, real b, int64_t n) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(n > 1 || (n == 1 && (a == b)), 3, "invalid number of points");
  if (THCTensor_(nElement)(state, r_) != n) THCTensor_(resize1d)(state, r_, n);
  if (n == 1) THCTensor_(fill)(state, r_, THCNumerics<real>::exp10(a));
  else {
    THCTensor *r = THCTensor_(isContiguous)(state, r_)
                   ? r_
                   : THCTensor_(newContiguous)(state, r_);
    real step = THCNumerics<real>::div(THCNumerics<real>::sub(b, a),
                                       ScalarConvert<int64_t,real>::to(n - 1));
    LogspaceOp<real> logspace_method(a, step);
    thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
    thrust::tabulate(data_, data_ + n, logspace_method);
    if (!THCTensor_(isContiguous)(state, r_)) {
      THCTensor_(freeCopyTo)(state, r, r_);
    }
  }
  THCudaCheck(cudaGetLastError());
}

#endif

void THCTensor_(range)(THCState *state, THCTensor *r_, accreal xmin, accreal xmax, accreal step) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, r_));
  THArgCheck(step > 0 || step < 0, 3, "step must be a non-null number");
  THArgCheck(((step > 0) && (xmax >= xmin)) || ((step < 0) && (xmax <= xmin))
              , 2, "upper bound and larger bound incoherent with step sign");
  ptrdiff_t size = (ptrdiff_t) (((xmax - xmin) / step) + 1);
  if (THCTensor_(nElement)(state, r_) != size) THCTensor_(resize1d)(state, r_, size);
  THCTensor *r = THCTensor_(isContiguous)(state, r_)
                 ? r_
                 : THCTensor_(newContiguous)(state, r_);
  LinspaceOp<real,accreal> linspace_method(xmin, step);
  thrust::device_ptr<real> data_(THCTensor_(data)(state, r));
  thrust::tabulate(data_, data_ + size, linspace_method);
  if (!THCTensor_(isContiguous)(state, r_)) THCTensor_(freeCopyTo)(state, r, r_);
  THCudaCheck(cudaGetLastError());
}

#endif
